#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <Windows.h>

#define	MAX(x,y) ((x)>(y)?(x):(y))
#define	MIN(x,y) ((x)<(y)?(x):(y))

hipError_t addWithCuda(BYTE* a, BYTE* b, unsigned int size, int val);

__global__ void addKernel(BYTE *a, BYTE *b, int max, int val)
{
	int pos = blockIdx.x * blockDim.x + threadIdx.x;
	if (pos >= max)
		return;
	auto v = MIN(255, a[pos]+val);
	b[pos] = MAX(0, v);
}

int mainN()
{
	BITMAPFILEHEADER hf;
	BITMAPINFOHEADER hinfo;
	RGBQUAD hRGB[256];
	FILE* fp;
	fp = fopen("l.bmp", "rb");
	if (fp == NULL)
		return -1;
	fread(&hf, sizeof(BITMAPFILEHEADER), 1, fp);
	fread(&hinfo, sizeof(BITMAPINFOHEADER), 1, fp);
	fread(hRGB, sizeof(RGBQUAD), 256, fp);
	int imgSize = hinfo.biWidth * hinfo.biHeight;

	BYTE* image = (BYTE*)malloc(imgSize);
	BYTE* output = (BYTE*)malloc(imgSize);

	fread(image, sizeof(BYTE), imgSize, fp);
	fclose(fp);

	addWithCuda(image, output, imgSize, -40);

	fp = fopen("o.bmp", "wb");
	fwrite(&hf, sizeof(BITMAPFILEHEADER), 1, fp);
	fwrite(&hinfo, sizeof(BITMAPINFOHEADER), 1, fp);
	fwrite(hRGB, sizeof(RGBQUAD), 256, fp);
	fwrite(output, sizeof(BYTE), imgSize, fp);
	fclose(fp);

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t addWithCuda(BYTE *a, BYTE *b, unsigned int size, int val)
{
	BYTE*dev_a = 0;
	BYTE*dev_b = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(BYTE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(BYTE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(BYTE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	int blocks = floor(size / 1024.0f + 0.5f);
    addKernel<<<blocks, 1024>>>(dev_a, dev_b, size, val);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(b, dev_b, size * sizeof(BYTE), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
